
#include <hip/hip_runtime.h>
#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU

#include "tensor2ipc.h"
#include "tensorflow/core/framework/op.h"
#include "tensorflow/core/framework/op_kernel.h"

using namespace tensorflow;

using GPUDevice = Eigen::GpuDevice;

// GPU specialization of actual computation.
//TODO Add cuda error checks here
template<typename T>
void TF2IPCFunctor<GPUDevice, T>::operator()(const GPUDevice& d, int size, void* address, const T* in, T** ipc_memory) {
    if(!(*ipc_memory)) {
      cudaIpcMemHandle_t* ipc_handle = reinterpret_cast<cudaIpcMemHandle_t*> (address);
      cudaIpcOpenMemHandle((void**) (ipc_memory), *ipc_handle, cudaIpcMemLazyEnablePeerAccess);
    }
    cudaMemcpy((void *) (*ipc_memory), (const void *) (in), size * sizeof(T), cudaMemcpyDeviceToDevice);
}

// Explicitly instantiate functors for the types of OpKernels registered.
template struct TF2IPCFunctor<GPUDevice, float>;
template struct TF2IPCFunctor<GPUDevice, double>;

#endif  // GOOGLE_CUDA
