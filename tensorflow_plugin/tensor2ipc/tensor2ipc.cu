
#include <hip/hip_runtime.h>
#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU

#include "tensor2ipc.h"
#include "tensorflow/core/framework/op.h"
#include "tensorflow/core/framework/op_kernel.h"

using namespace tensorflow;

using GPUDevice = Eigen::GpuDevice;

// GPU specialization of actual computation.
// TODO Add cuda error checks here
template <typename T>
void TF2IPCFunctor<GPUDevice, T>::operator()(const GPUDevice& d, int size,
                                             void* address, const T* in,
                                             T** ipc_memory) {
  cudaMemcpy((void*)(*ipc_memory), (const void*)(in), size * sizeof(T),
             cudaMemcpyDeviceToDevice);
}

// Explicitly instantiate functors for the types of OpKernels registered.
template struct TF2IPCFunctor<GPUDevice, float>;
template struct TF2IPCFunctor<GPUDevice, double>;

#endif  // GOOGLE_CUDA
