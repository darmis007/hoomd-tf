#include "hip/hip_runtime.h"
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "TensorflowCompute.cuh"


/*! \file TensorflowCompute.cu
    \brief CUDA kernels for TensorflowCompute
*/

// First, the kernel code for zeroing the velocities on the GPU
//! Kernel that zeroes velocities on the GPU
/*! \param d_vel Velocity-mass array from the ParticleData
    \param N Number of particles

    This kernel executes one thread per particle and zeros the velocity of each. It can be run with any 1D block size
    as long as block_size * num_blocks is >= the number of particles.
*/
extern "C" __global__
void gpu_add_scalar4_kernel(Scalar4 *dest, Scalar4 *src, unsigned int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
    {
        dest[i].x += src[i].x;
        dest[i].y += src[i].y;
        dest[i].z += src[i].z;
        dest[i].w += src[i].w;
    }
}

hipError_t gpu_add_scalar4(Scalar4 *dest, Scalar4 *src, unsigned int _N)
{
    // setup the grid to run the kernel
    int block_size = 256;
    dim3 grid( (int)ceil((double)_N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_add_scalar4_kernel<<< grid, threads >>>(dest, src, _N);

    // this method always succeds. If you had a cuda* call in this driver, you could return its error code if not
    // hipSuccess
    return hipSuccess;
}

extern "C" __global__
void gpu_add_virial_kernel(Scalar *dest, Scalar *src, unsigned int _N, unsigned int _pitch)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < _N)
    {
        dest[0 * _pitch + i] += src[i * 9 + 0]; //xx
        dest[1 * _pitch + i] += src[i * 9 + 1]; //xy
        dest[2 * _pitch + i] += src[i * 9 + 2]; //xz
        dest[3 * _pitch + i] += src[i * 9 + 4]; //yy
        dest[4 * _pitch + i] += src[i * 9 + 5]; //yz
        dest[5 * _pitch + i] += src[i * 9 + 8]; //zz
    }
}

hipError_t gpu_add_virial(Scalar *dest, Scalar *src, unsigned int _N, unsigned int _pitch)
{
    // setup the grid to run the kernel
    int block_size = 256;
    dim3 grid( (int)ceil((double)_N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_add_virial_kernel<<< grid, threads >>>(dest, src, _N, _pitch);

    // this method always succeds. If you had a cuda* call in this driver, you could return its error code if not
    // hipSuccess
    return hipSuccess;
}